
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void mult(char* s1, char* s2, int size){
    int i = threadIdx.x;
    s2[i] = s1[i % size];
}

int main(){
    hipMemcpyKind htd = hipMemcpyHostToDevice,
                   dth = hipMemcpyDeviceToHost;

    printf("Enter n: ");
    int n; scanf("%d", &n);

    printf("Enter string size: ");
    int size; scanf("%d", &size);

    int s = sizeof(char);
    char s1[size], s2[n * size], *d_s1, *d_s2;
    hipMalloc((void**)&d_s1, s * size);
    hipMalloc((void**)&d_s2, s * size * n);
    
    printf("Enter s1: ");
    for(int i = 0; i < size; i++) scanf(" %c", s1 + i);
    hipMemcpy(d_s1, s1, size * s, htd);

    mult<<<1, n * size>>>(d_s1, d_s2, size);
    hipMemcpy(s2, d_s2, n * size * s, dth);

    for(int i = 0; i < n * size; i++) 
        printf("%c", s2[i]);
    printf("\n");

}