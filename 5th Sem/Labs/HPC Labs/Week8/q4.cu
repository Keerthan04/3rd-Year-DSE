
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void reverse(char* s, char* rev, int len){
    int i = threadIdx.x;
    rev[i] = s[len - 1 - i];
}

int main(){
    int len, char_s = sizeof(char); 
    char s[100], *d_s, *d_rev;

    printf("Enter string: ");
    fgets(s, 100, stdin);
    for(len = 0; s[len] != '\0'; len++); len--;

    hipMalloc((void**)&d_s, char_s * len);
    hipMalloc((void**)&d_rev, char_s * len);

    hipMemcpy(d_s, s, len * char_s, hipMemcpyHostToDevice);
    reverse<<<1, len>>>(d_s, d_rev, len);
    hipMemcpy(s, d_rev, char_s * len, hipMemcpyDeviceToHost);

    puts(s);

}