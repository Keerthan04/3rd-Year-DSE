
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void word_reverse(char* s, char* rev, int words[]){
    int i = threadIdx.x;
    for(int x = words[i]; x < words[i + 1] - 1; x++)
        rev[x] = s[words[i + 1] - 2 - x + words[i]];
    rev[words[i + 1] - 1] = ' ';
}

int main(){
    int n, size = sizeof(char), i;
    printf("Number of words: ");
    scanf("%d", &n);

    int words[n + 1], *d_words; words[0] = 0;
    char s[100], *d_s, *d_rev;

    printf("Enter string: ");
    fgets(s, 100, stdin);
    fgets(s, 100, stdin);

    int t = 1;
    for(i = 0; s[i] != '\0'; i++)
        if(s[i] == ' ') words[t++] = i + 1;
    words[t] = i;

    hipMalloc((void**)&d_rev, size * words[t]);
    hipMalloc((void**)&d_s, size * words[t]);
    hipMalloc((void**)&d_words, (t + 1) * sizeof(int));

    hipMemcpy(d_s, s, size * words[t], hipMemcpyHostToDevice);
    hipMemcpy(d_words, words, sizeof(int) * (t + 1), hipMemcpyHostToDevice);

    word_reverse<<<1, t>>>(d_s, d_rev, d_words);
    hipMemcpy(s, d_rev, size * words[t], hipMemcpyDeviceToHost);

    puts(s);
}