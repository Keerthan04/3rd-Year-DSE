
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void word_count(char* s, char* sub, int* count, int words[], int subl){
    int i = threadIdx.x;
    if(words[i + 1] - 1 - words[i] != subl) return;
    for(int x = 0; x < subl; x++)
        if(s[words[i] + x] != sub[x]) return;
    atomicAdd(count, 1);
    
}
int main(){
    int n, size = sizeof(char), i, count = 0;
    int *d_count;
    printf("Number of words: ");
    scanf("%d", &n);

    int words[n + 1], *d_words; words[0] = 0;
    char s[100], sub[100], *d_s, *d_sub;

    printf("Enter string: ");
    fgets(s, 100, stdin);
    fgets(s, 100, stdin);

    int t = 1;
    for(i = 0; s[i] != '\0'; i++)
        if(s[i] == ' ') words[t++] = i + 1;
    words[t] = i;

    printf("Enter substring: ");
    fgets(sub, 100, stdin);
    int subl = 0; for(; sub[subl] != '\0'; subl++); subl--;

    hipMalloc((void**)&d_s, size * words[t]);
    hipMalloc((void**)&d_sub, size * subl);
    hipMalloc((void**)&d_words, (t + 1) * sizeof(int));
    hipMalloc(&d_count, sizeof(int));

    hipMemcpy(d_s, s, size * words[t], hipMemcpyHostToDevice);
    hipMemcpy(d_sub, sub, size * subl, hipMemcpyHostToDevice);
    hipMemcpy(d_words, words, sizeof(int) * (t + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);

    word_count<<<1, t>>>(d_s, d_sub, d_count, d_words, subl);
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    printf("%d\n", count);
    
}