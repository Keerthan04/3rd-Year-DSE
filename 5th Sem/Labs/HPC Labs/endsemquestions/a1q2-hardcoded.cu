#include "hip/hip_runtime.h"
/*
Cuda question... Input matrix n*n is taken... All principal diagonal elements are set to 0,upper half set to their factorial and lower half set to the sum of number.
sum of number clarification - 43 then ans is 7  (f bc)
*/
//tested with hard coded -> NO GPU

%%writefile q.cu
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <ctype.h>
#include<math.h>

#define N 3 // Matrix size for testing (N x N)

// Device function for calculating factorial
__device__ int factorial(int num) {
    if (num <= 1) return 1;
    return num * factorial(num - 1);
}

// Device function for calculating the sum of digits
__device__ int sum_of_digits(int num) {
    int sum = 0;
    while (num > 0) {
        sum += num % 10;
        num /= 10;
    }
    return sum;
}

// Kernel function for matrix calculation
__global__ void matrix_cal(int *matrix, int n) {
    int i = threadIdx.x;
    int j = threadIdx.y;

    if (i < n && j < n) {
        if (i == j) {
            matrix[i * n + j] = 0; // Set diagonal elements to 0
        } else if (i > j) {
            matrix[i * n + j] = sum_of_digits(matrix[i * n + j]); // Lower triangle: sum of digits
        } else {
            matrix[i * n + j] = factorial(matrix[i * n + j]); // Upper triangle: factorial
        }
    }
}

int main() {
    int matrix[N * N] = {2, 3, 4, 55, 6, 4, 12, 15, 10}; // Hardcoded 3x3 matrix

    int *d_matrix;
    int size = N * N * sizeof(int);

    // Allocate memory on the device
    hipMalloc((void**)&d_matrix, size);

    // Copy matrix from host to device
    hipMemcpy(d_matrix, matrix, size, hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 threadsPerBlock(N, N);

    // Launch kernel
    matrix_cal<<<1, threadsPerBlock>>>(d_matrix, N);

    // Copy result back to host
    hipMemcpy(matrix, d_matrix, size, hipMemcpyDeviceToHost);

    // Print the resulting matrix
    printf("The final matrix is:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", matrix[i * N + j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_matrix);

    return 0;
}
