/*
Cuda question... Input matrix n*n is taken... All principal diagonal elements are set to 0,upper half set to their factorial and lower half set to the sum of number.
sum of number clarification - 43 then ans is 7  (f bc)
*/
#include<stdio.h>
#include<stdlib.h>
#include<ctype.h>
#include<hip/hip_runtime.h>
#include<math.h>
//basic wala with 1 block only

//to call from device use of device functions(since global runs on kernel so)
__device__ int factorial(int num) {
    if (num <= 1) {
        return 1;
    }
    return num * factorial(num - 1);
}

__device__ int sum_of_digits(int num) {
    int sum = 0;
    while (num > 0) {
        sum += num % 10;
        num /= 10;
    }
    return sum;
}

__global__ void matrix_cal(int *matrix, int n) {
    int i = threadIdx.x;//since just 1 block used
    int j = threadIdx.y;
    
    if (i < n && j < n) {
        if (i == j) {
            matrix[i * n + j] = 0;  // Principal diagonal set to 0
        } else if (i > j) {
            matrix[i * n + j] = sum_of_digits(matrix[i * n + j]);  // Lower triangle - sum of digits
        } else {
            matrix[i * n + j] = factorial(matrix[i * n + j]);  // Upper triangle - factorial
        }
    }
}

int main() {
    int n;
    printf("Enter n for an n*n matrix: \n");
    scanf("%d", &n);

    int matrix_size = n * n;
    int matrix[matrix_size];

    printf("Enter the elements of the matrix:\n");
    for (int i = 0; i < matrix_size; i++) {
        scanf("%d", &matrix[i]);
    }

    int *d_matrix;
    int size = matrix_size * sizeof(int);

    // Allocate memory on the device
    hipMalloc((void**)&d_matrix, size);
    
    // Copy the matrix to the device
    hipMemcpy(d_matrix, matrix, size, hipMemcpyHostToDevice);

    // Set up a 2D grid with one block and n x n threads(basic one can do with variabl block but itna mehnat nahi)
    dim3 threadsPerBlock(n, n);
    matrix_cal<<<1, threadsPerBlock>>>(d_matrix, n);

    // Copy the modified matrix back to the host
    hipMemcpy(matrix, d_matrix, size, hipMemcpyDeviceToHost);

    // Print the modified matrix
    printf("The final matrix is:\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", matrix[i * n + j]);//use of row major
        }
        printf("\n");
    }


    hipFree(d_matrix);
    return 0;
}