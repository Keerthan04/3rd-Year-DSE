//take two matrix(n x n) and then add and multiply them(use 2d blocks and 2d threads)
#include<stdio.h>
#include<stdlib.h>
#include<ctype.h>
#include<hip/hip_runtime.h>
#include<math.h>


__global__ void matrix_cal(int *a,int *b,int *res,int *add, int m,int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;//since 2d block and 2d thread
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    if (i < m && j < n) {
        int val = 0;
        for (int k = 0; k < n; k++) {
            val += a[i * n + k] * b[k * n + j];
        }
        res[i * n + j] = val;
        add[i * n + j] = a[i * n + j] + b[i * n + j];
    }
}

int main() {
    int int_s = sizeof(int), m, n;
    printf("Enter m, n: ");
    scanf("%d %d", &m, &n);

    int a[m * n], b[m * n], res[m * n], add[m * n];
    printf("Enter a: ");
    for (int i = 0; i < m * n; i++) scanf("%d", a + i);
    printf("Enter b: ");
    for (int i = 0; i < m * n; i++) scanf("%d", b + i);

    int *d_a, *d_b, *d_res, *d_add;
    hipMalloc((void**)&d_a, int_s * m * n);
    hipMalloc((void**)&d_b, int_s * m * n);
    hipMalloc((void**)&d_res, int_s * m * n);
    hipMalloc((void**)&d_add, int_s * m * n);

    hipMemcpy(d_a, a, int_s * m * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, int_s * m * n, hipMemcpyHostToDevice);
    int blockSize = 2;//expecting multiple of 2 ka input(else just to ceil)
    dim3 dimBlock(2, 2);
    dim3 dimGrid(m / blockSize, n / blockSize);
    matrix_cal<<<dimGrid, dimBlock>>>(d_a, d_b, d_res, d_add, m, n);
    hipMemcpy(res, d_res, int_s * m * n, hipMemcpyDeviceToHost);
    hipMemcpy(add, d_add, int_s * m * n, hipMemcpyDeviceToHost);
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", res[n * i + j]);
        }
        printf("\n");
    }
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", add[n * i + j]);
        }
        printf("\n");
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);
    hipFree(d_add);
    return 0;
}