
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void inclusive_scan(int* a, int size){
    __shared__ int a_shared[4];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < size) a_shared[threadIdx.x] = a[i];
    __syncthreads();

    for(int x = i + 1; x < size; x++) 
        atomicAdd(&a[x], a_shared[i]);
}

int main(){
    int size;
    printf("Enter size: ");
    scanf("%d", &size);

    int a[size];
    printf("Enter array: ");
    for(int i = 0; i < size; i++) scanf("%d", a + i);

    int *d_a; hipMalloc(&d_a, sizeof(int) * size);
    hipMemcpy(d_a, a, sizeof(int) * size, hipMemcpyHostToDevice);

    inclusive_scan<<<ceil(size / 4.0), 4>>>(d_a, size);
    hipMemcpy(a, d_a, sizeof(int) * size, hipMemcpyDeviceToHost);

    for(int i = 0; i < size; i++) printf("%d ", a[i]);
    printf("\n");
}