
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void convolution(int* x, int* f, int* r, int n, int m){
    __shared__ int x_shared[4];
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < n) x_shared[threadIdx.x] = x[i];
    __syncthreads();

    for(int j = 0; j < m; j++)
        if(i + j < n) 
            atomicAdd(&r[i + j], x_shared[threadIdx.x] * f[m - 1 - j]);       
}

int main(){
    hipMemcpyKind htd = hipMemcpyHostToDevice,
                   dth = hipMemcpyDeviceToHost;

    int n, m, s = sizeof(int);
    printf("Enter n, m: ");
    scanf("%d %d", &n, &m);

    int x[n], f[m], r[n];
    printf("Enter array: ");
    for(int i = 0; i < n; i++){
        scanf("%d", x + i);
        r[i] = 0;
    }

    printf("Enter mask: ");
    for(int i = 0; i < m; i++)
        scanf("%d", f + i);

    int *d_x, *d_f, *d_r;
    hipMalloc((void**)&d_x, s * n);
    hipMalloc((void**)&d_f, s * m);
    hipMalloc((void**)&d_r, s * n);

    hipMemcpy(d_x, x, s * n, htd);
    hipMemcpy(d_f, f, s * m, htd);
    hipMemcpy(d_r, r, s * n, htd);
   
    convolution<<<ceil(n / 4.0), 4>>>(d_x, d_f, d_r, n, m);
    hipMemcpy(r, d_r, s * n, dth);

    for(int i = 0; i < n; i++)
        printf("%d ", r[i]);
    printf("\n");

}