
#include <hip/hip_runtime.h>
#include<stdio.h>
__constant__ int array[100], filter[20];
__global__ void convolution(int* r, int n, int m){

    int i = threadIdx.x, j = threadIdx.y;
    if(m-1 - j + i < n) atomicAdd(r + (m-1 - j + i), array[i] * filter[j]);
}

int main(){
    int n, m, s = sizeof(int);
    printf("Enter n, m: ");
    scanf("%d %d", &n, &m);

    int x[n], f[m], r[n];
    printf("Enter array: ");
    for(int i = 0; i < n; i++){
        scanf("%d", x + i);
        r[i] = 0;
    }

    printf("Enter mask: ");
    for(int i = 0; i < m; i++)
        scanf("%d", f + i);

    
    int *d_r;
    hipMemcpyToSymbol(HIP_SYMBOL(array), x, sizeof(float) * n);
    hipMemcpyToSymbol(HIP_SYMBOL(filter), f, sizeof(float) * m);
    hipMalloc((void**)&d_r, s * n);
    
    dim3 threads(n, m);
    convolution<<<1, threads>>>(d_r, n, m);
    hipMemcpy(r, d_r, s * n, hipMemcpyDeviceToHost);

    for(int i = 0; i < n; i++)
        printf("%d ", r[i]);
    printf("\n");

}