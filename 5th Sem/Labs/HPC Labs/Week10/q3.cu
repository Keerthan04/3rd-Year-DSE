
#include <hip/hip_runtime.h>
#include <stdio.h>
//Something wrong for sure + ChatGPT nonsense
__global__ void spmv_csr_kernel(int num_rows, float *values, int *col_indices, int *row_offsets, float *x, float *y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < num_rows) {
        float dot = 0.0f;
        int row_start = row_offsets[row];
        int row_end = row_offsets[row + 1];
        for (int jj = row_start; jj < row_end; jj++) {
            dot += values[jj] * x[col_indices[jj]];
        }
        y[row] = dot;
    }
}



int main() {
    float values[5] = {10, 20, 30, 40, 50};
    int col_indices[5] = {0, 2, 1, 0, 1};
    int row_offsets[4] = {0, 2, 3, 5};
    float x[3] = {1, 2, 3};
    int num_rows = 3;
    float y[3] = {0, 0, 0};

    float *d_values, *d_x, *d_y;
    int *d_col_indices, *d_row_offsets;
    int float_s = sizeof(float);
    int int_s = sizeof(int);

    hipMalloc(&d_values, 5 * float_s);
    hipMalloc(&d_col_indices, 5 * int_s);
    hipMalloc(&d_row_offsets, 4 * int_s);
    hipMalloc(&d_x, 3 * float_s);
    hipMalloc(&d_y, num_rows * float_s);

    hipMemcpy(d_values, values, 5 * float_s, hipMemcpyHostToDevice);
    hipMemcpy(d_col_indices, col_indices, 5 * int_s, hipMemcpyHostToDevice);
    hipMemcpy(d_row_offsets, row_offsets, 4 * int_s, hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, 3 * float_s, hipMemcpyHostToDevice);

    int blockSize = 4;
    int gridSize = ceil((float) num_rows / blockSize);
    spmv_csr_kernel<<<gridSize, blockSize>>>(num_rows, d_values, d_col_indices, d_row_offsets, d_x, d_y);

    hipMemcpy(y, d_y, num_rows * float_s, hipMemcpyDeviceToHost);

    printf("Result of SpMV (CSR): ");
    for (int i = 0; i < num_rows; i++) {
        printf("%f ", y[i]);
    }
    printf("\n");
}
