
#include <hip/hip_runtime.h>
#include<stdio.h>
//ChatGPT, should be correct

__global__ void conv2D(float *input, float *output, int x_m, int x_n, float *mask, int f) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float value = 0.0f;

    int pad = f / 2;

    for (int i = -pad; i <= pad; i++) 
        for (int j = -pad; j <= pad; j++) {

            int curRow = row + i;
            int curCol = col + j;

            if (curRow >= 0 && curRow < x_n && curCol >= 0 && curCol < x_m) 
                value += input[curRow * x_m + curCol] * mask[(i + pad) * f + (j + pad)];
            
        }

    if (row < x_n && col < x_m) {
        output[row * x_m + col] = value;
    }
}

int main() {
    int x_m, x_n, f, float_s = sizeof(float);
    
    printf("Enter input dimensions: ");
    scanf("%d %d", &x_m, &x_n);
    
    printf("Enter filter size: ");
    scanf("%d", &f);

    float h_input[x_m * x_n], h_mask[f * f], h_output[x_m * x_n];
    printf("Enter matrix: ");
    for(int i = 0; i < x_m * x_n; i++) 
        scanf("%f", h_input + i);
    
    printf("Enter filter: ");
    for(int i = 0; i < f * f; i++) 
        scanf("%f", h_mask + i);

    float *d_input, *d_output, *d_mask;
    hipMalloc(&d_input, x_m * x_n * float_s);
    hipMalloc(&d_output, x_m * x_n * float_s);
    hipMalloc(&d_mask, f * f * float_s);

    hipMemcpy(d_input, h_input, x_m * x_n * float_s, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, h_mask, f * f * float_s, hipMemcpyHostToDevice);

    dim3 dimBlock(2, 2);
    dim3 dimGrid(ceil(x_m / 2.0), ceil(x_n / 2.0));
    conv2D<<<dimGrid, dimBlock>>>(d_input, d_output, x_m, x_n, d_mask, f);
    hipMemcpy(h_output, d_output, x_m * x_n * float_s, hipMemcpyDeviceToHost);

    for (int i = 0; i < x_n; i++) {
        for (int j = 0; j < x_m; j++) {
            printf("%0.2f ", h_output[i * x_m + j]);
        }
        printf("\n");
    }
}
