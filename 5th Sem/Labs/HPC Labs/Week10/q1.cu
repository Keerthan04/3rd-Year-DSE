
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void mult(int *a, int *b, int *c, int ma, int na, int nb){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i >= ma || j >= nb) return;
    c[na * i + j] = 0;
    for(int k = 0; k < na; k++)
        atomicAdd(c + na * i + j, a[na * i + k] * b[nb * k + j]);
}

int main(){
    int int_s = sizeof(int), ma, na, mb, nb;
    printf("Enter m, n for a: ");
    scanf("%d %d", &ma, &na);

    printf("Enter m, n for b: ");
    scanf("%d %d", &mb, &nb);

    int a[ma * na], b[mb * nb], c[ma * nb];
    printf("Enter a: ");
    for(int i = 0; i < ma * na; i++) scanf("%d", a + i);
    printf("Enter b: ");
    for(int i = 0; i < mb * nb; i++) scanf("%d", b + i);

    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, int_s * ma * na);
    hipMalloc((void**)&d_b, int_s * mb * nb);
    hipMalloc((void**)&d_c, int_s * ma * nb);

    hipMemcpy(d_a, a, int_s * ma * na, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, int_s * mb * nb, hipMemcpyHostToDevice);

    mult<<<dim3(2, 2), dim3(ceil(ma/2.0), ceil(nb/2.0))>>>(d_a, d_b, d_c, ma, na, nb);
    hipMemcpy(c, d_c, int_s * ma * nb, hipMemcpyDeviceToHost);

    for(int i = 0; i < ma; i++){
        for(int j = 0; j < nb; j++)
            printf("%d ", c[nb * i + j]);
        printf("\n");
    }
}