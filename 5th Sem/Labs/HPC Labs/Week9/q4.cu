
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void complement(int *a, int m, int n){
    int i = threadIdx.x;
    if(i / m == 0 || i / m == n - 1 || i % n == 0 || i % n == n - 1) return;
    int pow2 = 1;
    while(pow2 <= a[i]) pow2 *= 2;
    int dec = a[i] ^ (pow2 - 1), bin = 0, offset = 1;
    while(dec > 0){
        bin += (dec % 2) * offset;
        offset *= 10;
        dec /= 2;
    }
    a[i] = bin;
}

int main(){
    int m, n;
    printf("Enter m, n: ");
    scanf("%d %d", &m, &n);

    int a[m * n];
    printf("Enter matrix: ");
    for(int i = 0; i < m * n; i++) scanf("%d", a + i);

    int *d_a;
    hipMalloc(&d_a, sizeof(int) * m * n);
    hipMemcpy(d_a, a, sizeof(int) * m * n, hipMemcpyHostToDevice);

    complement<<<1, m * n>>>(d_a, m, n);
    hipMemcpy(a, d_a, sizeof(int) * m * n, hipMemcpyDeviceToHost);
    for(int i = 0; i < m; i++){
        for(int j = 0; j < n; j++)
            printf("%d ", a[i * n + j]);
        printf("\n");
    }
    

}