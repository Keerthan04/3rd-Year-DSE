
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void row_mult(int *a, int *b, int *c, int ma, int na, int nb){
    int i = threadIdx.x;
    for(int j = 0; j < nb; j++){
        c[na * i + j] = 0;
        for(int k = 0; k < na; k++)
            atomicAdd(c + na * i + j, a[na * i + k] * b[nb * k + j]); 
    }
}
__global__ void col_mult(int *a, int *b, int *c, int ma, int na, int nb){
    int j = threadIdx.x;
    for(int i = 0; i < ma; i++){
        c[na * i + j] = 0;
        for(int k = 0; k < na; k++)
            atomicAdd(c + na * i + j, a[na * i + k] * b[nb * k + j]); 
    }
}
__global__ void ele_mult(int *a, int *b, int *c, int ma, int na, int nb){
    int i = threadIdx.x, j = threadIdx.y;
    c[na * i + j] = 0;
    for(int k = 0; k < na; k++)
        atomicAdd(c + na * i + j, a[na * i + k] * b[nb * k + j]);
}

int main(){
    int int_s = sizeof(int), ma, na, mb, nb;
    printf("Enter m, n for a: ");
    scanf("%d %d", &ma, &na);

    printf("Enter m, n for b: ");
    scanf("%d %d", &mb, &nb);

    int a[ma * na], b[mb * nb], c[ma * nb];
    printf("Enter a: ");
    for(int i = 0; i < ma * na; i++) scanf("%d", a + i);
    printf("Enter b: ");
    for(int i = 0; i < mb * nb; i++) scanf("%d", b + i);

    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, int_s * ma * na);
    hipMalloc((void**)&d_b, int_s * mb * nb);
    hipMalloc((void**)&d_c, int_s * ma * nb);

    hipMemcpy(d_a, a, int_s * ma * na, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, int_s * mb * nb, hipMemcpyHostToDevice);

    row_mult<<<1, ma>>>(d_a, d_b, d_c, ma, na, nb);
    
    hipMemcpy(c, d_c, int_s * ma * nb, hipMemcpyDeviceToHost);
    for(int i = 0; i < ma; i++){
        for(int j = 0; j < nb; j++)
            printf("%d ", c[nb * i + j]);
        printf("\n");
    }
    
    col_mult<<<1, nb>>>(d_a, d_b, d_c, ma, na, nb);
    
    hipMemcpy(c, d_c, int_s * ma * nb, hipMemcpyDeviceToHost);
    for(int i = 0; i < ma; i++){
        for(int j = 0; j < nb; j++)
            printf("%d ", c[nb * i + j]);
        printf("\n");
    }
    
    ele_mult<<<1, dim3(ma, nb)>>>(d_a, d_b, d_c, ma, na, nb);
    hipMemcpy(c, d_c, int_s * ma * nb, hipMemcpyDeviceToHost);

    for(int i = 0; i < ma; i++){
        for(int j = 0; j < nb; j++)
            printf("%d ", c[nb * i + j]);
        printf("\n");
    }
}