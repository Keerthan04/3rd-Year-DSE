
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void row_add(int *a, int *b, int *c, int m, int n){
    int i = threadIdx.x;
    for(int j = 0; j < n; j++) c[n * i + j] = a[n * i + j] + b[n * i + j];
}
__global__ void col_add(int *a, int *b, int *c, int m, int n){
    int j = threadIdx.x;
    for(int i = 0; i < m; i++) c[n * i + j] = a[n * i + j] + b[n * i + j];
}
__global__ void ele_add(int *a, int *b, int *c, int m, int n){
    int i = threadIdx.x, j = threadIdx.y;
    c[n * i + j] = a[n * i + j] + b[n * i + j];
}

int main(){
    int int_s = sizeof(int), m, n;
    printf("Enter m, n: ");
    scanf("%d %d", &m, &n);

    int a[m * n], b[m * n], c[m * n];
    printf("Enter a: ");
    for(int i = 0; i < m * n; i++) scanf("%d", a + i);
    printf("Enter b: ");
    for(int i = 0; i < m * n; i++) scanf("%d", b + i);

    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, int_s * m * n);
    hipMalloc((void**)&d_b, int_s * m * n);
    hipMalloc((void**)&d_c, int_s * m * n);

    hipMemcpy(d_a, a, int_s * m * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, int_s * m * n, hipMemcpyHostToDevice);

    row_add<<<1, m>>>(d_a, d_b, d_c, m, n);
    /*
    cudaMemcpy(c, d_c, int_s * m * n, cudaMemcpyDeviceToHost);
    for(int i = 0; i < m; i++){
        for(int j = 0; j < n; j++)
            printf("%d ", c[n * i + j]);
        printf("\n");
    }
    */
    col_add<<<1, n>>>(d_a, d_b, d_c, m, n);
    /*
    cudaMemcpy(c, d_c, int_s * m * n, cudaMemcpyDeviceToHost);
    for(int i = 0; i < m; i++){
        for(int j = 0; j < n; j++)
            printf("%d ", c[n * i + j]);
        printf("\n");
    }
    */
    ele_add<<<1, dim3(m, n)>>>(d_a, d_b, d_c, m, n);
    hipMemcpy(c, d_c, int_s * m * n, hipMemcpyDeviceToHost);

    for(int i = 0; i < m; i++){
        for(int j = 0; j < n; j++)
            printf("%d ", c[n * i + j]);
        printf("\n");
    }
}