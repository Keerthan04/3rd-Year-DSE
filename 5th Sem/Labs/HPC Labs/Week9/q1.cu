
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void power(int *a, int n){
    int i = threadIdx.x, j = threadIdx.y;
    int res = a[n * i + j];
    for(int x = 0; x < i; x++) a[n * i + j] *= res;
}

int main(){
    int m, n, int_s = sizeof(int);
    printf("Enter m, n: ");
    scanf("%d %d", &m, &n);

    int a[m * n], *d_a;
    printf("Enter matrix: ");
    for(int i = 0; i < m * n; i++) 
        scanf("%d", a + i);

    hipMalloc((void**)&d_a, int_s * m * n);
    hipMemcpy(d_a, a, int_s * m * n, hipMemcpyHostToDevice);

    power<<<1, dim3(m, n)>>>(d_a, n);
    hipMemcpy(a, d_a, int_s * m * n, hipMemcpyDeviceToHost);
    
    for(int i = 0; i < m; i++){
        for(int j = 0; j < n; j++)
            printf("%d ", a[n * i + j]);
        printf("\n");
    }
}