
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void linear(int a, int* x, int* y){
    int i = threadIdx.x;
    y[i] += a * x[i];
}

int main(){
    hipMemcpyKind htd = hipMemcpyHostToDevice, 
                   dth = hipMemcpyDeviceToHost;

    printf("Enter size: ");
    int n; scanf("%d", &n);

    int x[n], y[n], a, *d_x, *d_y, size = sizeof(int);
    hipMalloc((void**)&d_x, n * size);
    hipMalloc((void**)&d_y, n * size);

    printf("Enter x: ");
    for(int i = 0; i < n; i++) scanf("%d", x + i);
    hipMemcpy(d_x, x, size * n, htd);

    printf("Enter y: ");
    for(int i = 0; i < n; i++) scanf("%d", y + i);
    hipMemcpy(d_y, y, size * n, htd);

    printf("Enter a: ");
    scanf("%d", &a);

    linear<<<1, n>>>(a, d_x, d_y);
    hipMemcpy(y, d_y, size * n, dth);

    for(int i = 0; i < n; i++)
        printf("%d ", y[i]);
    printf("\n");

}