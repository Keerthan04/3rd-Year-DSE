
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void convolution(int* x, int* f, int* r, int n, int m){
    int i = threadIdx.x, j = threadIdx.y;
    if(m-1 - j + i < n) atomicAdd(r + (m-1 - j + i), x[i] * f[j]);
}

int main(){
    hipMemcpyKind htd = hipMemcpyHostToDevice,
                   dth = hipMemcpyDeviceToHost;

    int n, m, s = sizeof(int);
    printf("Enter n, m: ");
    scanf("%d %d", &n, &m);

    int x[n], f[m], r[n];
    printf("Enter array: ");
    for(int i = 0; i < n; i++){
        scanf("%d", x + i);
        r[i] = 0;
    }

    printf("Enter mask: ");
    for(int i = 0; i < m; i++)
        scanf("%d", f + i);

    
    int *d_x, *d_f, *d_r;
    
    hipMalloc((void**)&d_x, s * n);
    hipMalloc((void**)&d_f, s * m);
    hipMalloc((void**)&d_r, s * n);
    
    hipMemcpy(d_x, x, s * n, htd);
    hipMemcpy(d_f, f, s * m, htd);
    hipMemcpy(d_r, r, s * n, htd);
    
    dim3 threads(n, m, 1);
    convolution<<<1, threads>>>(d_x, d_f, d_r, n, m);

    hipMemcpy(r, d_r, s * n, dth);
    for(int i = 0; i < n; i++)
        printf("%d ", r[i]);
    printf("\n");

}
