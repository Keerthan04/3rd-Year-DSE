
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void compute_sine(float* angle, float* sine){
    int i = threadIdx.x;
    sine[i] = sin(angle[i]);
}

int main(){
    int n;
    printf("Enter size: ");
    scanf("%d", &n);

    float angle[n], sine[n], *d_angle, *d_sine;
    hipMalloc((void**)&d_angle, sizeof(float) * n);
    hipMalloc((void**)&d_sine, sizeof(float) * n);

    printf("Enter array: ");
    for(int i = 0; i < n; i++) scanf("%f", angle + i);
    hipMemcpy(d_angle, angle, sizeof(float) * n, hipMemcpyHostToDevice);

    compute_sine<<<1, n>>>(d_angle, d_sine);
    hipMemcpy(sine, d_sine, sizeof(float) * n, hipMemcpyDeviceToHost);

    for(int i = 0; i < n; i++) printf("%f ", sine[i]);
    printf("\n");

    hipFree(d_angle);
    hipFree(d_sine);
}