
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void block_add(float *out, float *a, float *b) {
    int i = blockIdx.x;
    out[i] = a[i] + b[i];     
}

__global__ void thread_add(float *out, float *a, float *b) {
    int i = threadIdx.x;
    out[i] = a[i] + b[i];     
}

__global__ void var_add(float *out, float *a, float *b, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n) out[i] = a[i] + b[i];     
}

int main(){
    hipMemcpyKind htd = hipMemcpyHostToDevice, dth = hipMemcpyDeviceToHost;
    int n;
    printf("Enter size: ");
    scanf("%d", &n);
    
    float a[n], b[n], out[n];
    float *da, *db, *dout;

    hipMalloc((void**)&da, sizeof(float) * n);
    hipMalloc((void**)&db, sizeof(float) * n);
    hipMalloc((void**)&dout, sizeof(float) * n);

    printf("Enter a: ");
    for(int i = 0; i < n; i++)
        scanf("%f", a + i);
    
    printf("Enter b: ");
    for(int i = 0; i < n; i++)
        scanf("%f", b + i);

    hipMemcpy(da, a, sizeof(float) * n, htd);
    hipMemcpy(db, b, sizeof(float) * n, htd);
    hipMemcpy(dout, out, sizeof(float) * n, htd);

    block_add<<<n, 1>>>(dout, da, db);
    hipMemcpy(out, dout, n * sizeof(float), dth);

    for(int i = 0; i < n; i++)
        printf("%f ", out[i]);
    printf("\n");

    thread_add<<<1, n>>>(dout, da, db);
    hipMemcpy(out, dout, n * sizeof(float), dth);
    
    for(int i = 0; i < n; i++)
        printf("%f ", out[i]);
    printf("\n");

    var_add<<<ceil(n/256.0), 256>>>(dout, da, db, n);
    hipMemcpy(out, dout, n * sizeof(float), dth);
    
    for(int i = 0; i < n; i++)
        printf("%f ", out[i]);
    printf("\n");
    
    hipFree(da);
    hipFree(db);
    hipFree(dout);

}